#include "hip/hip_runtime.h"

#include "parzendataterm.h"
#include "errorhandling.h"
#include "config.h"
#include "utils.h"

#include <limits.h>
#include <float.h>


// constant used in computaion of gauss kernel
#define SQRT_2PI 2.50662827463

// small epsilon to avoid log(0)
#define LOG0_EPSILON 0.0000001f

// constants indication special cases
#define LIKELIHOOD_FIXED_THIS_LABEL (-0.0001f) // pick negative value close to 0 so visualization of normalized probability is not off
#define LIKELIHOOD_FIXED_OTHER_LABEL (-0.0002f) // pick negative value close to 0 so visualization of normalized probability is not off
#define ENERGY_FIXED_OTHER_LABEL 1000


namespace tvseg {
namespace cuda {


using std::cout;
using std::cerr;
using std::endl;


// TODO: optimize this by precomputing parts of the expression
__device__ float kernel_gauss(const float x, const float var)
{
    return expf(-0.5f*x*x/(var*var))/(var*SQRT_2PI);
}


// given dataterm likelihood, compute the dataterm energy for one label as the negative log, taking into account special cases
__global__ void kernel_datatermEnergy(float * const dataterm, const int pitch, const Dim2 dim, const float min, const float max, const bool energyFixedOtherLabelUseMin)
{
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    if (x >= dim.width || y >= dim.height)
        return;

    const uint idx = x + y*pitch;

    const float prob = dataterm[idx];

    if (prob == LIKELIHOOD_FIXED_THIS_LABEL) {
        // -1  =>  scribble from current label on current pixel  =>  very low energy
        dataterm[idx] = 0;
    } else if(prob == LIKELIHOOD_FIXED_OTHER_LABEL) {
        // -2  =>  indicates scribble from different label on current pixel  =>  very high energy
        if (energyFixedOtherLabelUseMin) {
            dataterm[idx] = -log(LOG0_EPSILON);
        } else {
            dataterm[idx] = ENERGY_FIXED_OTHER_LABEL;
        }
    } else {
        if (min != max) {
            // dataterm contains likelihood which is >= 0. Normalization converts this to
            // range ]0,1], for which taking the negative log gives energies in the range [0, +inf[
            dataterm[idx] = -log((prob - min) / (max - min) * (1 - LOG0_EPSILON) + LOG0_EPSILON);
        } else {
            // in general min != max, since otherwise the likelihood for all pixels and all labels is the same,
            // i.e. no meaningful result is possible; handle it anyway to be safe
            dataterm[idx] = -log(prob + LOG0_EPSILON);
        }
    }
}



// get minimum and maximum value of dataterm likelihood, ignoring negative values (which have special meaning)
__global__ void kernel_likelihoodColumnsMinMax(const float * const dataterm, const int pitch, const Dim2 dim, float * const columnsMin, float * const columnsMax)
{
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;

    if (x >= dim.width)
        return;

    float min = FLT_MAX;
    float max = -FLT_MAX;

    for (int y = 0; y < dim.height; ++y)
    {
        const float value = dataterm[x + y*pitch];
        if (value >= 0) { // ignore negative values as they indicate special cases
            if (value < min) {
                min = value;
            }
            if (value > max) {
                max = value;
            }
        }
    }
    columnsMin[x] = min;
    columnsMax[x] = max;
}


__global__ void kernel_datatermLikelihood(
        float * const dataterm,
        float * const rhos,
        const float * const color,
        const float * const depth,
        const uint pitch, // dataterm, rhos, and depth
        const uint pitch3, // color
        const int * const scribbles,
        const int * const numScribbles,
        const float depthNormalization,
        const Intrinsics intr,
        const Dim3 dim,
        const ParzenParams params)
{

    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;
    if (x >= dim.width || y >= dim.height)
        return;

    const uint datatermImageSize = dim.height * pitch;

    const float depthScale = params.depthScale / depthNormalization;
    float xyScale = 1.0;
    if (params.normalizeSpaceScale) {
        if (params.scribbleDistancePerspective) {
            xyScale = depthScale;
        } else {
            xyScale = 1.0f / max(dim.width, dim.height);
        }
    }

    const float alpha = params.scribbleDistanceFactor;
    const float sigma = params.colorVariance;
    const float tau = params.depthVariance;

    // compute nearest neighbor scribbles
    int currentNumScribbles;
    for (uint i = 0, scribbleOffset = 0; i < dim.labels; ++i, scribbleOffset += currentNumScribbles*2) {
        currentNumScribbles = numScribbles[i];
        float rho = 0;

        if (params.useDistanceKernel) {
            // find distance to nearest neighbor scribble of this label
            float minDist = FLT_MAX;
            for (int n = scribbleOffset; n < scribbleOffset + currentNumScribbles*2; n += 2) {
                const int sx = scribbles[n];
                const int sy = scribbles[n + 1];
                float dx, dy, dz;
                if (params.scribbleDistanceDepth) {
                    if (params.scribbleDistancePerspective) {
                        // backproject pixel positions to 3d space using depth and camera intrinsics
                        const float fz = depth[x + y*pitch];
                        const float fsz = depth[sx + sy*pitch];
                        const float fx = fz*(x - intr.o_x) / intr.s_x;
                        const float fy = fz*(y - intr.o_y) / intr.s_y;
                        const float fsx = fsz*(sx - intr.o_x) / intr.s_x;
                        const float fsy = fsz*(sy - intr.o_y) / intr.s_y;
                        dx = fsx - fx;
                        dy = fsy - fy;
                        dz = fsz - fz;
                    } else {
                        // isometric projection
                        const float fz = depth[x + y*pitch];
                        const float fsz = depth[sx + sy*pitch];
                        dx = sx - (int)x;
                        dy = sy - (int)y;
                        dz = fsz - fz;
                    }
                } else {
                    dx = sx - (int)x;
                    dy = sy - (int)y;
                    dz = 0;
                }
                dx *= xyScale;
                dy *= xyScale;
                dz *= depthScale;
                float dist = dx*dx + dy*dy + dz*dz;
                if (dist < minDist) {
                    minDist = dist;
                }
            }
            if (params.fixScribblePixels && !params.fixScribblePixelsThisOnly && minDist == 0) {
                // there must be scribble of current label on this pixel  ==>  set hard constraints for all labels
                for (int j = 0; j < dim.labels; ++j) {
                    rhos[x + y*pitch + j*datatermImageSize] = 0; // use 0 for all labels here as this is only used for visualization
                    if (i == j) {
                        dataterm[x + y*pitch + j*datatermImageSize] = LIKELIHOOD_FIXED_THIS_LABEL;
                    } else {
                        dataterm[x + y*pitch + j*datatermImageSize] = LIKELIHOOD_FIXED_OTHER_LABEL;
                    }
                }
                return; // nothing else to do when hard constraints are set
            }
            // when there are no scribbles, minDist == FLT_MAX  ==>  leave rho as default value for sensible visualization
            if (minDist < FLT_MAX) {
                // when alpha == 0, indicating disabled space kernel, rho will also be 0
                rho = alpha * sqrtf(minDist);
            }
        }
        rhos[x + y*pitch + i*datatermImageSize] = rho;
    }

    for (uint i = 0, scribbleOffset = 0; i < dim.labels; scribbleOffset += currentNumScribbles*2, ++i)  {
        const float rho = rhos[x + y*pitch + i*datatermImageSize];
        currentNumScribbles = numScribbles[i];
        int numActiveScribbles = 0;
        bool fixedThisLabel = false;
        float likely_a = 0; // active scribbles close to the pixel
        float likely_i = 0; // inactive pixels further away
        if (currentNumScribbles > 0) {
            for (int n = scribbleOffset; n < scribbleOffset + currentNumScribbles*2 && !fixedThisLabel; n += 2) {
                bool active = true;
                float space_factor = 1.0;
                float color_factor = 1.0;
                float depth_factor = 1.0;
                const int sx = scribbles[n];
                const int sy = scribbles[n + 1];

                // space kernel
                if (params.useDistanceKernel && rho > 0) {
                    float dx, dy, dz;
                    if (params.scribbleDistanceDepth) {
                        if (params.scribbleDistancePerspective) {
                            // backproject pixel positions to 3d space using depth and camera intrinsics
                            const float fz = depth[x + y*pitch];
                            const float fsz = depth[sx + sy*pitch];
                            const float fx = fz*(x - intr.o_x) / intr.s_x;
                            const float fy = fz*(y - intr.o_y) / intr.s_y;
                            const float fsx = fsz*(sx - intr.o_x) / intr.s_x;
                            const float fsy = fsz*(sy - intr.o_y) / intr.s_y;
                            dx = fsx - fx;
                            dy = fsy - fy;
                            dz = fsz - fz;
                        } else {
                            // isometric projection
                            const float fz = depth[x + y*pitch];
                            const float fsz = depth[sx + sy*pitch];
                            dx = sx - (int)x;
                            dy = sy - (int)y;
                            dz = fsz - fz;
                        }
                    } else {
                        dx = sx - (int)x;
                        dy = sy - (int)y;
                        dz = 0;
                    }
                    dx *= xyScale;
                    dy *= xyScale;
                    dz *= depthScale;
                    float dist = sqrtf(dx*dx + dy*dy + dz*dz);
                    if (params.useSpatiallyActiveScribbles) {
                        if (dist <= (rho * params.spatialActivationFactor) / alpha) {
                            ++numActiveScribbles;
                        } else {
                            active = false;
                        }
                    } else {
                        // all scribbles active
                        ++numActiveScribbles;
                    }

                    space_factor = kernel_gauss(dist, rho);
                } else if (params.useDistanceKernel && rho == 0 && alpha != 0 &&
                           params.fixScribblePixels && params.fixScribblePixelsThisOnly) {
                    dataterm[x + y*pitch + i*datatermImageSize] = LIKELIHOOD_FIXED_THIS_LABEL;
                    fixedThisLabel = true;
                }

                // color kernel
                if (params.useColorKernel && sigma > 0)
                {
                    // Euclidean distance in RGB color space
                    // TODO: is rgb really in that order for us? (doesn't really matter for euclidean distance)
                    const float r = color[3*x + y*pitch3    ] - color[3*sx + sy*pitch3    ];
                    const float g = color[3*x + y*pitch3 + 1] - color[3*sx + sy*pitch3 + 1];
                    const float b = color[3*x + y*pitch3 + 2] - color[3*sx + sy*pitch3 + 2];
                    const float dist = sqrtf(r*r + g*g + b*b);
                    color_factor = kernel_gauss(dist * params.colorScale, sigma);
                }

                // depth kernel
                if (params.useDepthKernel && tau > 0)
                {
                    const float d = depth[x + y*pitch] - depth[sx + sy*pitch];
                    depth_factor = kernel_gauss(d * depthScale, tau);
                }

                if (active) {
                    likely_a += space_factor * color_factor * depth_factor;
                } else {
                    likely_i += space_factor * color_factor * depth_factor;
                }
            }
            const int numInactiveScribbles = currentNumScribbles - numActiveScribbles;
//            if (x == 0 && y == 0) {
//                printf("currentNumScribbles: %d, activescribbles: %d, numinactivescribbles: %d\n", currentNumScribbles, numActiveScribbles, numInactiveScribbles);
//            }
//            if (x == 0 && y == 0) {
//                printf("likely_a: %f, likely_i: %f\n", likely_a, likely_i);
//            }
            if (params.useSpatiallyActiveScribbles && (((float)numActiveScribbles) / currentNumScribbles) < params.activeScribbleLambda && numActiveScribbles > 0) {
                // if active scribbles have a smaller portion than params.activeScribbleLambda, make sure they have overall weight of params.activeScribbleLambda
                likely_a /= numActiveScribbles;
                if (numInactiveScribbles > 0) {
                    likely_i /= numInactiveScribbles;
                }
            } else {
                // if active scribbles already have a larger portion than params.activeScribbleLambda, don't adjust weight at all
                likely_a = (likely_a + likely_i) / currentNumScribbles;
                likely_i = likely_a;
            }
//            if (x == 0 && y == 0) {
//                printf("likely_a: %f, likely_i: %f\n", likely_a, likely_i);
//            }
        }
        if (fixedThisLabel) {
            dataterm[x + y*pitch + i*datatermImageSize] = LIKELIHOOD_FIXED_THIS_LABEL;
        } else {
            dataterm[x + y*pitch + i*datatermImageSize] = likely_a * params.activeScribbleLambda + likely_i * (1 - params.activeScribbleLambda);
        }
    }
}


float normalizationDepthScale(const float * const depth, const Dim2 dim, const bool perspective, const Intrinsics &intr) {
    if (dim.width == 0 || dim.height == 0) {
        return 1.0;
    }
    float minx = FLT_MAX;
    float miny = FLT_MAX;
    float minz = FLT_MAX;
    float maxx = -FLT_MAX;
    float maxy = -FLT_MAX;
    float maxz = -FLT_MAX;
    for (uint x = 0; x < dim.width; ++x) {
        for (uint y = 0; y < dim.height; ++y) {
            const float fz = depth[x + y*dim.width];
            const float fx = fz*(x - intr.o_x) / intr.s_x;
            const float fy = fz*(y - intr.o_y) / intr.s_y;
            minx = min(fx, minx);
            miny = min(fy, miny);
            minz = min(fz, minz);
            maxx = max(fx, maxx);
            maxy = max(fy, maxy);
            maxz = max(fz, maxz);
        }
    }
    const float dx = maxx - minx;
    const float dy = maxy - miny;
    const float dz = maxz - minz;
    float scale;
    if (perspective) {
        scale = max(dx, max(dy, dz));
    } else {
        scale = dz;
    }
    if (scale == 0) {
        scale = 1;
    }
    return scale;
}


void computeParzenDataterm(
        float* dataterm,
        const float *color,
        const float *depth,
        const int *scribbles,
        const int *numScribbles,
        const Intrinsics &intr,
        const Dim3 &dim,
        const ParzenParams &params,
        const Feedback *feedback)
{
    // debugging info
    cout << "Begin: computeParzenDataterm" << endl;
    cout << "\tdim: " << dim.width << "," << dim.height << "," << dim.labels << endl;
    cout << "\tparams: " << params.colorScale << "," << params.depthScale << "," << params.colorVariance << "," << params.depthVariance << "," << params.useColorKernel << "," << params.useDepthKernel << "," << params.fixScribblePixels << "," << params.useDistanceKernel << ","  << params.normalizeSpaceScale << "," << params.activeScribbleLambda << endl;
    cout << "\tscribble counts: ";
    for (int i = 0; i < dim.labels; ++i) {
        cout << numScribbles[i] << ", ";
    }
    cout << endl;
//    cout << "\tscribbles: " << endl;
//    uint currentNumScribbles;
//    for (uint i = 0, scribbleOffset = 0; i < dim.labels; i++, scribbleOffset += currentNumScribbles*2) {
//        currentNumScribbles = numScribbles[i];
//        cout << "\t\t" << i+1 << " : ";
//        for (int n = scribbleOffset; n < scribbleOffset + currentNumScribbles*2; n += 2) {
//            cout << "(" << scribbles[n] << "," << scribbles[n+1] << ")  ";
//        }
//        cout << endl;
//    }

    // Prepare grid size
    dim3 dimBlock(BLOCKDIMX, BLOCKDIMY);
    dim3 dimGrid;
    dimGrid.x = (dim.width % dimBlock.x) ? (dim.width/dimBlock.x + 1) : (dim.width/dimBlock.x);
    dimGrid.y = (dim.height % dimBlock.y) ? (dim.height/dimBlock.y + 1) : (dim.height/dimBlock.y);

    // auxiliare variables
    size_t pitch; // pitch for dataterm and depth
    size_t pitch3; // pitch for color
    int totalNumScribbles = 0;
    for (int i = 0; i < dim.labels; ++i) {
        int num = numScribbles[i];
        if (num < 0 || num > dim.width * dim.height) {
            cerr << "unreasonable scribble count " << num << "... aborting" << endl;
            return;
        }
        totalNumScribbles += num;
    }
    cout << "\ttotalNumScribbles: " << totalNumScribbles << endl;
    int steps = 0; // for feedback
    const int maxSteps = 4 + dim.labels;
    feedback->progress(steps++, maxSteps, 0);

    // normalize depth
    float depthNormalization = 1.0;
    if (params.normalizeSpaceScale) {
//        bool perspectiveNormalization = params.scribbleDistancePerspective;
        bool perspectiveNormalization = false;
        depthNormalization = normalizationDepthScale(depth, dim2(dim), perspectiveNormalization, intr);
    }

    // /////////////////////////////////////////////////////////////////
    // 1. allocate memory on the GPU and copy data or set initial value

    float *gpu_dataterm = NULL;
    float *gpu_rhos = NULL;
    float *gpu_color = NULL;
    float *gpu_depth = NULL;
    int *gpu_scribbles = NULL;
    int *gpu_numScribbles = NULL;
    float *gpu_columnsMin = NULL;
    float *gpu_columnsMax = NULL;

    const uint widthBytes = dim.width * sizeof(float);
    const uint heightTotal = dim.height * dim.labels;

    CUDA_CHECK(hipMallocPitch((void**)&gpu_dataterm, &pitch, widthBytes, heightTotal));
    CUDA_CHECK(hipMemset2DAsync(gpu_dataterm, pitch, 0, widthBytes, heightTotal));

    CUDA_CHECK(hipMallocPitch((void**)&gpu_rhos, &pitch, widthBytes, heightTotal));
    CUDA_CHECK(hipMemset2DAsync(gpu_rhos, pitch, 0, widthBytes, heightTotal));

    CUDA_CHECK(hipMallocPitch((void**)&gpu_color, &pitch3, 3*widthBytes, dim.height));
    CUDA_CHECK(hipMemcpy2DAsync(gpu_color, pitch3, color, 3*widthBytes, 3*widthBytes, dim.height, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMallocPitch((void**)&gpu_depth, &pitch, widthBytes, dim.height));
    CUDA_CHECK(hipMemcpy2DAsync(gpu_depth, pitch, depth, widthBytes, widthBytes, dim.height, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void**)&gpu_scribbles, 2 * totalNumScribbles * sizeof(int)));
    CUDA_CHECK(hipMemcpyAsync(gpu_scribbles, scribbles, 2 * totalNumScribbles * sizeof(int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc((void**)&gpu_numScribbles, dim.labels * sizeof(int)));
    CUDA_CHECK(hipMemcpyAsync(gpu_numScribbles, numScribbles, dim.labels * sizeof(int), hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&gpu_columnsMin, widthBytes));
    CUDA_CHECK(hipMalloc(&gpu_columnsMax, widthBytes));

    feedback->progress(steps++, maxSteps, 0);

    // /////////////////////////////////////////////////////////////////
    // 2.  compute dataterm likelihood on GPU
    CUDA_CHECK_KERNEL_LAUNCH_AND_SYNCHRONIZE();
    kernel_datatermLikelihood<<< dimGrid, dimBlock >>>(gpu_dataterm, gpu_rhos, gpu_color, gpu_depth,
                                                       pitch/sizeof(float), pitch3/sizeof(float),
                                                       gpu_scribbles, gpu_numScribbles,
                                                       depthNormalization, intr, dim, params);
    CUDA_CHECK_KERNEL_LAUNCH_AND_SYNCHRONIZE();
    std::string winName = "rhos";
    CUDA_CHECK(hipMemcpy2D((void*)dataterm, widthBytes, gpu_rhos, pitch, widthBytes, heightTotal, hipMemcpyDeviceToHost));
    feedback->displayImage(dataterm, dim, winName);
    winName = "likely";
    CUDA_CHECK(hipMemcpy2D((void*)dataterm, widthBytes, gpu_dataterm, pitch, widthBytes, heightTotal, hipMemcpyDeviceToHost));
    feedback->displayImage(dataterm, dim, winName);

    feedback->progress(steps++, maxSteps, 0);

    // /////////////////////////////////////////////////////////////////
    // 3.  compute min/mix of likelihood for normalization of likelihood values

    dim3 dimBlockMinMax(BLOCKDIMX * BLOCKDIMY, 1);
    dim3 dimGridMinMax;
    dimGridMinMax.x = (dim.width % dimBlockMinMax.x) ? (dim.width / dimBlockMinMax.x + 1) : (dim.width / dimBlockMinMax.x);
    dimGridMinMax.y = 1;

    float *columnsMin = new float[dim.width];
    float *columnsMax = new float[dim.width];
    float min = FLT_MAX;
    float max = -FLT_MAX;

    // TODO: maybe compute for the long columns of all labels with one kernel call?
    for (int i = 0; i < dim.labels; ++i) {
        if (feedback->isCanceled()) {
            break;
        }
        kernel_likelihoodColumnsMinMax<<< dimGridMinMax, dimBlockMinMax >>>(gpu_dataterm + i * pitch/sizeof(float) * dim.height,
                                                                            pitch/sizeof(float),
                                                                            dim2(dim),
                                                                            gpu_columnsMin,
                                                                            gpu_columnsMax);
        CUDA_CHECK_KERNEL_LAUNCH_AND_SYNCHRONIZE();
        CUDA_CHECK(hipMemcpy(columnsMin, gpu_columnsMin, widthBytes, hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(columnsMax, gpu_columnsMax, widthBytes, hipMemcpyDeviceToHost));
        for (int j = 0; j < dim.width; ++j) {
            if (columnsMin[j] < min) {
                min = columnsMin[j];
            }
            if (columnsMax[j] > max) {
                max = columnsMax[j];
            }
        }
        feedback->progress(steps++, maxSteps, 0);
    }
    delete [] columnsMin;
    delete [] columnsMax;

//    cout << "likelihood range: [" << min << "," << max << "]" << endl;

    // /////////////////////////////////////////////////////////////////
    // 4.  compute energy from likelihood
    for (uint i = 0; i < dim.labels; ++i) {
        if (feedback->isCanceled()) {
            break;
        }
        kernel_datatermEnergy<<< dimGrid, dimBlock >>>(gpu_dataterm + i * pitch/sizeof(float) * dim.height,
                                                       pitch/sizeof(float),
                                                       dim2(dim), min, max,
                                                       params.fixScribblePixelsOtherLabelUseMin);
        CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();
        feedback->progress(steps++, maxSteps, 0);
    }

    // copy result back to CPU
    if (!feedback->isCanceled()) {
        CUDA_CHECK(hipMemcpy2D((void*)dataterm, dim.width * sizeof(float), gpu_dataterm, pitch, dim.width * sizeof(float), dim.height * dim.labels, hipMemcpyDeviceToHost));
        feedback->progress(steps++, maxSteps, 0);
    }

    // delete all data on GPU
    CUDA_CHECK(hipFree(gpu_dataterm));
    CUDA_CHECK(hipFree(gpu_rhos));
    CUDA_CHECK(hipFree(gpu_color));
    CUDA_CHECK(hipFree(gpu_depth));
    CUDA_CHECK(hipFree(gpu_scribbles));
    CUDA_CHECK(hipFree(gpu_numScribbles));
}


}
}
