#include "hip/hip_runtime.h"

#include "solver.h"
#include "errorhandling.h"
#include "config.h"
#include "cvimageshow.h"
#include "utils.h"

#include <nppi.h>

#include <iostream>
#include <time.h>



namespace tvseg {
namespace cuda {


using std::cout;
using std::endl;


// FIXME: use debug logging provided

// FIXME: clean up (memory) on stack unwind


// get the difference for each pixel between u and u_saved
__global__ void kernel_getPixelDifferences(
        float *pixelDifferences,
        const float *u,
        const float *u_saved,
        const Dim3 dim,
        const uint indexPitch)
{
    // which thread are we?
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    // is it within image?
    if (x >= dim.width || y >= dim.height)
        return;

    const uint idx = x + y * indexPitch;
    const uint labelOffset = indexPitch * dim.height;

    pixelDifferences[idx] = 0.0f;

    for (uint i = 0; i < dim.labels; ++i) {
        // add up pixel differences
        // NOTE: Since the changes for a single pixel sum up to zero, each change is counted
        //       twice. We take care of this when summing up pixelDifferences.
        pixelDifferences[idx] += abs(u[idx + i * labelOffset] - u_saved[idx + i * labelOffset]);
    }
}



// get energy of a solution
__global__ void kernel_getEnergy(
        float *energy,
        const float *u,
        const float *g,
        const float *dataterm,
        const float lambda,
        const Dim3 dim,
        const uint indexPitch
        )
{
    // which thread are we?
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    // is it within image?
    if(x >= dim.width || y >= dim.height)
        return;

    const uint idx = x + y * indexPitch;
    const uint labelOffset = indexPitch * dim.height;
    const float g_scaled = lambda * g[idx];

    // accumulator for energy (energy_xy = energy[x + y * pitch]); summing over all labels
    float energy_xy_acc = 0.0f;

    for (int i = 0; i < dim.labels; ++i) {
        const int pos_u = idx + i * labelOffset;
        const float u_pos_u = u[pos_u];

        // compute norm of gradient of u
        const float grad_u_x = (x + 1 < dim.width ) ? (u[pos_u + 1         ] - u_pos_u) : 0.0f;
        const float grad_u_y = (y + 1 < dim.height) ? (u[pos_u + indexPitch] - u_pos_u) : 0.0f;
        const float norm_grad_u = sqrtf(grad_u_x * grad_u_x + grad_u_y * grad_u_y);

        // determine value of standard energy ( |gradU| = - divXi  )
        energy_xy_acc += dataterm[pos_u] * u_pos_u + g_scaled * norm_grad_u;
    }

    energy[idx] = energy_xy_acc;
}



// get thresholded u (to show it)
__global__ void kernel_getUThresh(
        float *u_thresh,
        const float *u,
        const Dim3 dim,
        const uint indexPitch)
{
    // which thread are we?
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    // is it within image?
    if(x >= dim.width || y >= dim.height)
        return;

    const uint idx = x + y * indexPitch;
    const uint labelOffset = indexPitch * dim.height;

    // compute maximum and index of maximum
    // we set all labels to 0.0, except the maximum, which is set to 1.0
    int pos_u_max = idx;
    float u_max = u[idx];
    u_thresh[idx] = 0.0f;

    for (int i = 1; i < dim.labels; ++i) {
        const uint pos_u  = idx + i * labelOffset;
        u_thresh[pos_u] = 0.0f;
        const float u_pos_u = u[pos_u];
        if (u_max < u_pos_u) {
            pos_u_max = pos_u;
            u_max = u_pos_u;
        }
    }

    u_thresh[pos_u_max] = 1.0f;
}



// get the segmentation as image of label indices
__global__ void kernel_getSegmentation(
        unsigned char *segmentation,
        const float *u_thresh,
        Dim3 dim,
        const int indexPitchUChar,
        const int indexPitchFloat)
{
    // which thread are we?
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    // is it within image?
    if(x >= dim.width || y >= dim.height)
        return;

    const uint idxFloat = x + y * indexPitchFloat;
    const uint idxUChar = x + y * indexPitchUChar;
    const uint labelOffset = indexPitchFloat * dim.height;

    // find index where thresholded u == 1.0 (should be exactly one)
    unsigned char index = 0;
    for (int i = 1; i < dim.labels; ++i) {
        if (1.0f == u_thresh[idxFloat + i * labelOffset]) {
            index = i;
        }
    }

    segmentation[idxUChar] = index;
}



// calculate gradient ascent = update dual variables
__global__ void kernel_grad_ascent(
        float *xi,                // dual variables
        float *psi,               // dual variables
        const float *u_overrelax, // overrelaxed primal variables
        const float *sum_u,       // current pixelwise sum of u; is optimized to be 1.0
        const float *g,           // weight for penailzing region jumps
        const float lambda,       // parameter for smoothness vs correctness
        const Dim3 dim,     // image dimensions
        const int indexPitch      // image pitch in indices (not bytes)
        )
{
    // which thread are we?
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    // is it within image?
    if(x >= dim.width || y >= dim.height)
        return;

    const uint idx = x + y * indexPitch;
    const uint labelOffset = indexPitch * dim.height;

    // step sizes tau
    const float scaleTau = 2.0f;
    const float tau_xi   = 1.0f / (2          * scaleTau);
    const float tau_psi  = 1.0f / (dim.labels * scaleTau);

    // combine g and lambda
    const float g_scaled = g[idx] * lambda;

    // update xi for each label
    for (uint i = 0; i < dim.labels; ++i) {

        // set positions within arrays
        const uint pos_u  = idx + i * labelOffset;
        const uint pos_xi = idx + i * labelOffset * 2;

        // cache values from global memory
        const float u_overrelax_pos_u = u_overrelax[pos_u];
        float xi_x = xi[pos_xi];
        float xi_y = xi[pos_xi + labelOffset];

        // update xi   (gradient as forward difference)
        // ================================================
        // compute:   xi  =  xi - tau_xi * grad(u_overrelax)
        //            project xi

        // FIMXE: Why not -= in the following??
        // TODO: Does higher-order discrete differences make sense?

        // for (x == dim.width - 1) we always have xi_x == 0.0 from initialization
        if (x + 1 < dim.width) {
            xi_x += tau_xi * (u_overrelax[pos_u + 1]          - u_overrelax_pos_u);
        }

        // for (y == dim.height - 1) we always have xi_y == 0.0 from initialization
        if (y + 1 < dim.height) {
            xi_y += tau_xi * (u_overrelax[pos_u + indexPitch] - u_overrelax_pos_u);
        }

        // project xi
        const float norm_xi = sqrtf(xi_x*xi_x + xi_y*xi_y) / g_scaled;
        if (norm_xi > 1.0f) {
            xi_x /= norm_xi;
            xi_y /= norm_xi;
        }

        // write xi to global memory
        xi[pos_xi] = xi_x;
        xi[pos_xi + labelOffset] = xi_y;
    }

    // update psi
    // ================================================
    // compute:   psi = psi + tau_psi * (sum_u - 1)    where sum_u is determined by using u_overrelax
    psi[idx] += tau_psi * (sum_u[idx] - 1.0f);

}


// calculate gradient descent = update primal variables
__global__ void kernel_grad_descent(
        float *u,
        float *u_overrelax,
        float *sum_u,
        const float *xi,
        const float *psi,
        const float *dataterm,
        const Dim3 dim,
        const uint indexPitch)
{
    // which thread are we?
    const uint x = blockDim.x * blockIdx.x + threadIdx.x;
    const uint y = blockDim.y * blockIdx.y + threadIdx.y;

    // is it within image?
    if(x >= dim.width || y >= dim.height)
        return;

    const uint idx = x + y * indexPitch;
    const uint labelOffset = indexPitch * dim.height;

    // compute step width
    const float scaleTau  = 2.0f;
    const float tau_u     = (scaleTau * 1.0f)/6.0f;

    // accumulator for sum_u
    float sum_u_tmp = 0.0f;

    // cache values from global memory
    const float psi_idx = psi[idx];

    // update u for each label
    for (uint i = 0; i < dim.labels; ++i) {

        // set positions within arrays
        const uint pos_u = idx + i * labelOffset;
        const uint pos_xi = idx + i * labelOffset * 2;

        // compute div_xi   (backward differences)
        // ================================================
        float div_xi;
        if (x == 0 && y == 0)
            div_xi = 0;
        else if (x == 0)
            div_xi = xi[pos_xi + labelOffset] - xi[pos_xi + labelOffset - indexPitch];
        else if (y == 0)
            div_xi = xi[pos_xi]               - xi[pos_xi               - 1];
        else {
            // divXi = backwDiff_x + backwDiff_y
            div_xi = xi[pos_xi]               - xi[pos_xi               - 1]
                   + xi[pos_xi + labelOffset] - xi[pos_xi + labelOffset - indexPitch];
        }

        // update u
        // ================================================
        // compute:   u = u - tau_u * (dataterm + div_xi + 1/numLabels * psi)
        //            project u
        //            extrapolate u

        // cache values from global memory
        float u_pos_u = u[pos_u];
        const float u_pos_u_old = u_pos_u;

        // FIXME: plus or minus?? also check order of div_xi computation
        //u_pos_u -= tau_u * (dataTerm[pos_u] + divXi + 1.0f/nRegions * psi_temp);
        u_pos_u -= tau_u * (dataterm[pos_u] - div_xi + 1.0f/dim.labels * psi_idx);

        // project u
        if(u_pos_u > 1.0f)
           u_pos_u = 1.0f;
        if(u_pos_u < 0.0f)
           u_pos_u = 0.0f;

        // determine overrelaxed u and accumulate sum_u
        const float u_overrelax_pos_u = 2.0f * u_pos_u - u_pos_u_old;
        sum_u_tmp += u_overrelax_pos_u;

        // commit to global memory
        u[pos_u]           = u_pos_u;
        u_overrelax[pos_u] = u_overrelax_pos_u;
    }

    // commit accumulated sum to global memory
    sum_u[idx] = sum_u_tmp;
}



void computeSolution(
        unsigned char *segmentation,
        float *u,
        const float *dataterm,
        const float *weight,
        const Dim3 &dim,
        const SolverParams &params,
        const Feedback *feedback,
        SolverOutput &output
        )
{
    //FIXME: Check input for =! NULL


    cout << "Begin: computeSolution" << endl;

    cout << "\tdim: " << dim.width << "," << dim.height << "," << dim.labels << endl;

    cout << "\tparams: " << params.lambda << "," << params.epsilon << "," << params.maxSteps << "," << params.minSteps << "," << params.autoRecompute << "," << params.displayIteration << "," << params.checkConverganceEvery << "," << params.displayIterationEvery << endl;

    ///////////////////////////////////////////////////////////////////////////
    /// 1) Prepare grid size
    ///////////////////////////////////////////////////////////////////////////
    dim3 dimBlock(BLOCKDIMX, BLOCKDIMY);
    dim3 dimGrid;
    size_t pitch;
    size_t pitchUChar;
    dimGrid.x = (dim.width % dimBlock.x) ? (dim.width/dimBlock.x + 1) : (dim.width/dimBlock.x);
    dimGrid.y = (dim.height % dimBlock.y) ? (dim.height/dimBlock.y + 1) : (dim.height/dimBlock.y);

    ///////////////////////////////////////////////////////////////////////////
    /// 2) allocate the memory on the GPU & copy the data there
    ///////////////////////////////////////////////////////////////////////////
    float *gpu_dataterm = NULL;
    float *gpu_g = NULL;
    float *gpu_u = NULL;
    float *gpu_u_overrelax = NULL;
    float *gpu_u_thresh = NULL;
    float *gpu_u_saved = NULL;
    float *gpu_sum_u = NULL;
    float *gpu_pixelDifferences = NULL;
    float *gpu_xi = NULL;
    float *gpu_psi = NULL;
    unsigned char *gpu_segmentation = NULL;
    float *gpu_energy_relax = NULL;
    float *gpu_energy_thresh = NULL;

    const uint widthBytes = dim.width * sizeof(float);
    const uint heightTotal = dim.height * dim.labels;
    const NppiSize nppiSize = { dim.width, dim.height };

    // ========================================================================
    // Input
    CUDA_CHECK(hipMallocPitch((void**)&gpu_dataterm, &pitch, widthBytes, heightTotal));             // gpu_dataterm
    CUDA_CHECK(hipMemcpy2D(gpu_dataterm, pitch, dataterm, widthBytes, widthBytes, heightTotal, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMallocPitch((void**)&gpu_g, &pitch, widthBytes, dim.height));                     // gpu_g
    CUDA_CHECK(hipMemcpy2D(gpu_g, pitch, weight, widthBytes, widthBytes, dim.height, hipMemcpyHostToDevice));

    // ========================================================================
    // Primal variables and helpers for primal-dual algo
    // FIXME: check that host data is sensible?
    // initialize u with supplied host data
    CUDA_CHECK(hipMallocPitch((void**)&gpu_u, &pitch, widthBytes, heightTotal));                    // gpu_u
    CUDA_CHECK(hipMemcpy2D(gpu_u, pitch, u, widthBytes, widthBytes, heightTotal, hipMemcpyHostToDevice));

    // FIXME: is initialization the right thing (tm) ???
    // initialize u_overrelax with u
    CUDA_CHECK(hipMallocPitch((void**)&gpu_u_overrelax, &pitch, widthBytes, heightTotal));          // gpu_u_overrelax
    CUDA_CHECK(hipMemcpy2D(gpu_u_overrelax, pitch, gpu_u, pitch, widthBytes, heightTotal, hipMemcpyDeviceToDevice));

    // FIXME: should initialize with actual sum??
    // initialize sum_u with 1
    CUDA_CHECK(hipMallocPitch((void**)&gpu_sum_u, &pitch, widthBytes, dim.height));                 // gpu_sum_u
    CUDA_CHECK(nppiSet_32f_C1R(1.0f, gpu_sum_u, pitch, nppiSize));

    // no need to initialize gpu_u_thresh; is set before first use
    CUDA_CHECK(hipMallocPitch((void**)&gpu_u_thresh, &pitch, widthBytes, heightTotal));             // gpu_u_thresh

    // ========================================================================
    // Dual Variables
    // FIXME: initialization with 0 sensible for xi and psi???
    CUDA_CHECK(hipMallocPitch((void**)&gpu_xi, &pitch, widthBytes, heightTotal * 2));               // gpu_xi
    CUDA_CHECK(hipMemset2D(gpu_xi, pitch, 0, widthBytes, heightTotal * 2));

    CUDA_CHECK(hipMallocPitch((void**)&gpu_psi, &pitch, widthBytes, dim.height));                   // gpu_psi
    CUDA_CHECK(hipMemset2D(gpu_psi, pitch, 0, widthBytes, dim.height));

    // ========================================================================
    // Helpers for checking abort criteria
    // initialize u_saved with 0, assuming that this is different from the initialization of u
    CUDA_CHECK(hipMallocPitch((void**)&gpu_u_saved, &pitch, widthBytes, heightTotal));              // gpu_u_saved
    CUDA_CHECK(hipMemset2D(gpu_u_saved, pitch, 0, widthBytes, heightTotal));

    // no need to initialize pixelDifferences; is set before first use
    CUDA_CHECK(hipMallocPitch((void**)&gpu_pixelDifferences, &pitch, widthBytes, dim.height));      // gpu_pixelDifferences

    // ========================================================================
    // Helpers to determine the energy gap
    // no need to initialize energies; are set before first use
    CUDA_CHECK(hipMallocPitch((void**)&gpu_energy_relax , &pitch, widthBytes, dim.height));         // gpu_energy_relax
    CUDA_CHECK(hipMallocPitch((void**)&gpu_energy_thresh, &pitch, widthBytes, dim.height));         // gpu_energy_thresh

    // ========================================================================
    // Output segmentation
    // no need to initialize; is set before first use
    CUDA_CHECK(hipMallocPitch((void**)&gpu_segmentation, &pitchUChar, dim.width, dim.height));    // gpu_segmentation

    ///////////////////////////////////////////////////////////////////////////
    /// 3) Additional Parameters
    ///////////////////////////////////////////////////////////////////////////
    // FIXME: what should this be?
    //float breakValue = 5e-06;     // mein Vorschlag: 0.0001f;  Evgeny: 0.001f;  => 0.0005f;
    //const double epsilon = 5e-4;

    ///////////////////////////////////////////////////////////////////////////
    /// 4) Helper variables
    ///////////////////////////////////////////////////////////////////////////

    Npp8u *gpu_nppDeviceBuffer = NULL;
    int nppBufferSize = 0;
    nppiSumGetBufferHostSize_32f_C1R(nppiSize, &nppBufferSize);
    CUDA_CHECK(hipMalloc((void **)(&gpu_nppDeviceBuffer), nppBufferSize));
    Npp64f *gpu_nppSum = NULL;
    CUDA_CHECK(hipMalloc((void **)(&gpu_nppSum), sizeof(Npp64f) * 1));

    // helper to display thresholded u
    float *u_thresh = new float [dim.width * heightTotal];

    // set output steps; overwritten when we abort before reaching maxSteps
    output.steps = params.maxSteps;
    output.cancelled = false;

    // start timing
    const clock_t timeStart = clock();

    ///////////////////////////////////////////////////////////////////////////
    /// 5) Run the primal-dual aglorithm iterations
    ///////////////////////////////////////////////////////////////////////////

    feedback->progress(0, params.maxSteps, 0);
    for (uint step = 1; step <= params.maxSteps; ++step) {

        // ===============================================================
        // Check for cancel
        // ===============================================================

        if (feedback->isCanceled()) {
            cout << "cancelled" << endl;
            output.cancelled = true;
            break;
        }

        // ===============================================================
        // Check abort criteria
        // ===============================================================
        if(step % params.checkConverganceEvery == 0) {
            cout << "step = " << step << "\t\t";

            // get the difference between the current u and the former iteration (u_saved)
            // also assigns u_saved to the current u
            kernel_getPixelDifferences<<< dimGrid, dimBlock >>>(gpu_pixelDifferences, gpu_u, gpu_u_saved, dim, pitch/sizeof(float));
            CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();

            // Sum up the pixel differences and divide by the number of pixels.
            // Since each change has to affect 2 regions equally (the sum of all regions remains 1),
            // we also have to divide by 2 to only count each change once.
            Npp64f change;
            CUDA_CHECK(nppiSum_32f_C1R(gpu_pixelDifferences, pitch, nppiSize, gpu_nppDeviceBuffer, gpu_nppSum));
            CUDA_CHECK(hipMemcpy(&change, gpu_nppSum, sizeof(Npp64f), hipMemcpyDeviceToHost));
            change /= 2.0 * dim.width * dim.height;
            cout << "change = " << change << endl;

            // Small change => close to convergance => stop
            if (change < params.epsilon && step >= params.minSteps) {
                cout << "Change is less than epsilon (" << params.epsilon << "). Abort." << endl;
                output.steps = step;
                break;
            }
        }

        // Save the current value of u in u_saved
        CUDA_CHECK(hipMemcpy2D((void*)gpu_u_saved, pitch, gpu_u, pitch, widthBytes, heightTotal, hipMemcpyDeviceToDevice));

        // ===============================================================
        // Updates of dual and primal variables
        // ===============================================================

        // update xi, psi
        kernel_grad_ascent<<< dimGrid, dimBlock >>>(gpu_xi, gpu_psi, gpu_u_overrelax, gpu_sum_u, gpu_g, params.lambda, dim, pitch/sizeof(float));
        CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();

        // update u
        kernel_grad_descent<<< dimGrid, dimBlock >>>(gpu_u, gpu_u_overrelax, gpu_sum_u, gpu_xi, gpu_psi, gpu_dataterm, dim, pitch/sizeof(float));
        CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();

        // ===============================================================
        // Display current segmentation
        // ===============================================================
        if (params.displayIteration && (step % params.displayIterationEvery == 0)) {

            // dispaly u
            std::string winName = "u";
            CUDA_CHECK(hipMemcpy2D((void*)u, widthBytes, gpu_u, pitch, widthBytes, heightTotal, hipMemcpyDeviceToHost));
            feedback->displayImage(u, dim, winName);
            //cvImageShow(u, dim.height, dim.width, dim.labels, winName.c_str());

            // get uThresh
            kernel_getUThresh<<< dimGrid, dimBlock >>>(gpu_u_thresh, gpu_u, dim, pitch/sizeof(float));
            CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();

            // dispaly thresholded u
            winName = "u_thresh";
            CUDA_CHECK(hipMemcpy2D((void*)u_thresh, widthBytes, gpu_u_thresh, pitch, widthBytes, heightTotal, hipMemcpyDeviceToHost));
            feedback->displayImage(u_thresh, dim, winName);
        }

        feedback->progress(step, params.maxSteps, 0);
    }
    output.time = (clock() - timeStart)/(float)CLOCKS_PER_SEC;

    cout << "took " << output.time << "s to solve with " << output.steps << "iterations" << endl;

    ///////////////////////////////////////////////////////////////////////////
    /// 6) determine energy gap
    ///////////////////////////////////////////////////////////////////////////

    // threshold u
    kernel_getUThresh<<< dimGrid, dimBlock >>>(gpu_u_thresh, gpu_u, dim, pitch/sizeof(float));
    CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();

    // determine energy for u
    Npp64f energy_relax;
    kernel_getEnergy<<< dimGrid, dimBlock >>>(gpu_energy_relax, gpu_u, gpu_g, gpu_dataterm, params.lambda, dim, pitch/sizeof(float));
    CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();
    CUDA_CHECK(nppiSum_32f_C1R(gpu_energy_relax, pitch, nppiSize, gpu_nppDeviceBuffer, gpu_nppSum));
    CUDA_CHECK(hipMemcpy(&energy_relax, gpu_nppSum, sizeof(Npp64f), hipMemcpyDeviceToHost));

    // determine energy for uThresh (energyThresh)
    Npp64f energy_thresh;
    kernel_getEnergy<<< dimGrid, dimBlock >>>(gpu_energy_thresh, gpu_u_thresh, gpu_g, gpu_dataterm, params.lambda, dim, pitch/sizeof(float));
    CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();
    CUDA_CHECK(nppiSum_32f_C1R(gpu_energy_thresh, pitch, nppiSize, gpu_nppDeviceBuffer, gpu_nppSum));
    CUDA_CHECK(hipMemcpy(&energy_thresh, gpu_nppSum, sizeof(Npp64f), hipMemcpyDeviceToHost));

    cout << "energy_thresh = " << energy_thresh << " and energy_relax = " << energy_relax << endl;
    output.energyGap = abs((energy_thresh - energy_relax)/energy_relax);
    cout << "energy_gap = " << output.energyGap << endl;

    ///////////////////////////////////////////////////////////////////////////
    /// 7) Compute segementation and copy result
    ///////////////////////////////////////////////////////////////////////////

    kernel_getSegmentation<<< dimGrid, dimBlock >>>(gpu_segmentation, gpu_u_thresh, dim, pitchUChar, pitch/sizeof(float));
    CUDA_CHECK_KERNEL_LAUNCH_AND_DEBUG_SYNCHRONIZE();

    // copy result back to CPU
    CUDA_CHECK(hipMemcpy2D((void*)u, widthBytes, gpu_u, pitch, widthBytes, heightTotal, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy2D((void*)segmentation, dim.width, gpu_segmentation, pitchUChar, dim.width, dim.height, hipMemcpyDeviceToHost));

    ///////////////////////////////////////////////////////////////////////////
    /// 8) clean up
    ///////////////////////////////////////////////////////////////////////////

    // free device memory
    CUDA_CHECK(hipFree(gpu_dataterm));
    CUDA_CHECK(hipFree(gpu_g));
    CUDA_CHECK(hipFree(gpu_u));
    CUDA_CHECK(hipFree(gpu_u_overrelax));
    CUDA_CHECK(hipFree(gpu_u_thresh));
    CUDA_CHECK(hipFree(gpu_u_saved));
    CUDA_CHECK(hipFree(gpu_sum_u));
    CUDA_CHECK(hipFree(gpu_pixelDifferences));
    CUDA_CHECK(hipFree(gpu_xi));
    CUDA_CHECK(hipFree(gpu_psi));
    CUDA_CHECK(hipFree(gpu_segmentation));
    CUDA_CHECK(hipFree(gpu_energy_relax));
    CUDA_CHECK(hipFree(gpu_energy_thresh));
    CUDA_CHECK(hipFree(gpu_nppDeviceBuffer));
    CUDA_CHECK(hipFree(gpu_nppSum));

    // free host memory
    delete u_thresh;

    cout << "End: computeSolution" << endl;
}


} // namespace cuda
} // namespace tvseg
